#include "hip/hip_runtime.h"
#define GLUT_DISABLE_ATEXIT_HACK

#include "kernel.cuh"
#include <stdlib.h>
#include <stdio.h>

// ---- Linux
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include "interactions.cuh"


// --- Texture and Pixel objects
GLuint pbo = 0;     // OpenGL pixel buffer object
GLuint tex = 0;     // OpenGL texture object
struct hipGraphicsResource *cuda_pbo_resource;

void render()
{
    uchar4 *d_out = 0;
    hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **) &d_out, NULL,
                                         cuda_pbo_resource);

    kernelLauncher(d_out, W, H, loc);
    hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}


void drawTexture()
{
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, W, H, 0, GL_RGBA,
                 GL_UNSIGNED_BYTE, NULL);
    glEnable(GL_TEXTURE_2D);
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(0, 0);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(0, H);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(W, H);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(W, 0);

    glEnd();
    glDisable(GL_TEXTURE_2D);
}

void display()
{
    render();
    drawTexture();
    glutSwapBuffers();
}

void initGLUT(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(W, H);
    glutCreateWindow(TITLE_STRING);

# ifndef __APPLE__
    glewInit();
#endif
}

void initPixelBuffer()
{
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 4*W*H*sizeof(GLubyte), 0, GL_STREAM_DRAW);
    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void exitfunc()
{
    if (pbo) {
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &pbo);
        glDeleteTextures(1, &tex);
    }
}

/* -----------------------------------------------
 *              M A I N
 * -----------------------------------------------
 */

int main(int argc, char **argv) {

    // --- Start Up
    printInstructions();
    initGLUT(&argc, argv);
    gluOrtho2D(0, W, H, 0);

    // --- User Controls : Keyboard
    glutKeyboardFunc(keyboard);
    glutSpecialFunc(handleSpecialKeypress);

    // --- User Controls : Mouse
    glutPassiveMotionFunc(mouseMove);
    glutMotionFunc(mouseDrag);
    glutDisplayFunc(display);

    // --- Begin
    initPixelBuffer();
    glutMainLoop();

    // --- End
    atexit(exitfunc);

    return 0;

}
